#include "utilities.h"

void print_model(Model h_model, int input_size, int num_classes)
{
    FILE *file = fopen("./matrixTests/test.txt", "w");
    if (file == NULL)
    {
        printf("Error opening file for writing.\n");
        return;
    }

    // Write the dimensions (rows and columns) at the beginning of the file
    fprintf(file, "Rows: %d, Columns: %d\n\n", input_size, num_classes);

    // Write the weights
    fprintf(file, "Model Weights:\n");
    for (int i = 0; i < num_classes; i++)
    {
        fprintf(file, "Class %d: ", i);
        for (int j = 0; j < input_size; j++)
        {
            fprintf(file, "%.6f ", h_model.weights[i * input_size + j]);
        }
        fprintf(file, "\n");
    }

    // Write the biases
    fprintf(file, "\nModel Biases:\n");
    for (int i = 0; i < num_classes; i++)
    {
        fprintf(file, "Bias for Class %d: %.6f\n", i, h_model.biases[i]);
    }

    // Close the file
    fclose(file);
}

int main()
{
    const char *image_file = "./dataSet/train-images.idx3-ubyte";
    const char *label_file = "./dataSet/train-labels.idx1-ubyte";

    int image_count, label_count, rows, cols;
    unsigned char *images = read_idx3_file(image_file, &image_count, &rows, &cols);

    unsigned char *labels = read_idx1_file(label_file, &label_count);
    if (image_count != label_count)
    {
        fprintf(stderr, "Image and label counts do not match!\n");
        return EXIT_FAILURE;
    }

    Model h_model;
    int input_size = rows * cols;
    h_model.weights = (float *)malloc(input_size * NUM_CLASSES * sizeof(float));
    h_model.biases = (float *)malloc(NUM_CLASSES * sizeof(float));

    Model d_model = init_model(input_size, NUM_CLASSES);
    float learning_rate = 0.1;

    float *d_images, *d_logits, *d_delta, *dt_images, *dt_deltas;
    bool *d_label;

    hipMalloc(&d_images, NUM_IMAGES * IMG_SIZE * sizeof(float));
    hipMalloc(&d_logits, NUM_IMAGES * NUM_CLASSES * sizeof(float));
    hipMalloc(&d_delta, NUM_IMAGES * NUM_CLASSES * sizeof(float));
    hipMalloc(&d_label, NUM_IMAGES * NUM_CLASSES * sizeof(bool));
    hipMalloc(&dt_images, NUM_IMAGES * IMG_SIZE * sizeof(float));
    hipMalloc(&dt_deltas, NUM_IMAGES * NUM_CLASSES * sizeof(float));

    // Copy data to GPU
    hipMemcpy(d_images, images, NUM_IMAGES * IMG_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_label, labels, NUM_CLASSES * NUM_IMAGES * sizeof(bool), hipMemcpyHostToDevice);
    hipMemset(d_logits, 0, NUM_IMAGES * NUM_CLASSES * sizeof(float));

    float *d_prob = d_logits;

    // Updated thread and block names
    dim3 threadsPerBlock_z(COMPUTE_Z_BLOCK_SIZE);
    dim3 blocksPerGrid_z(NUM_CLASSES * NUM_IMAGES);
    compute_z<<<blocksPerGrid_z, threadsPerBlock_z>>>(h_model.weights, h_model.biases, d_images, d_logits, IMG_SIZE, NUM_CLASSES, NUM_IMAGES);

    dim3 threadsPerBlock_softmax(NUM_CLASSES, SMAX_BLOCK_SIZE / NUM_CLASSES);
    dim3 blocksPerGrid_softmax(NUM_IMAGES * NUM_CLASSES / SMAX_BLOCK_SIZE);
    compute_softmax<<<blocksPerGrid_softmax, threadsPerBlock_softmax>>>(d_logits, d_prob, NUM_CLASSES * NUM_IMAGES);

    dim3 threadsPerBlock_subtract(32, 16);
    dim3 blocksPerGrid_subtract((NUM_IMAGES + threadsPerBlock_subtract.x - 1) / threadsPerBlock_subtract.x,
                                (NUM_CLASSES + threadsPerBlock_subtract.y - 1) / threadsPerBlock_subtract.y);
    matrixSubtractKernel<<<blocksPerGrid_subtract, threadsPerBlock_subtract>>>(d_prob, d_label, d_delta, NUM_IMAGES, NUM_CLASSES);

    dim3 threadsPerBlock_transpose(8, 16);
    dim3 blocksPerGrid_transpose((NUM_IMAGES + 8 - 1) / 8, (IMG_SIZE + 16 - 1) / 16);
    transpose<<<blocksPerGrid_transpose, threadsPerBlock_transpose>>>(d_images, dt_images, NUM_IMAGES, NUM_CLASSES);

    dim3 threadsPerBlock_transpose_prob(8, 16);
    dim3 blocksPerGrid_transpose_prob((NUM_IMAGES + 8 - 1) / 8, (NUM_CLASSES + 16 - 1) / 16);
    transpose<<<blocksPerGrid_transpose_prob, threadsPerBlock_transpose_prob>>>(d_prob, dt_deltas, NUM_IMAGES, NUM_CLASSES);

    dim3 threadsPerBlock_update_biases(NUM_IMAGES);
    dim3 blocksPerGrid_update_biases(NUM_CLASSES);
    update_biases<<<blocksPerGrid_update_biases, threadsPerBlock_update_biases, IMG_SIZE * sizeof(float)>>>(dt_deltas, d_model.biases, learning_rate, NUM_CLASSES, IMG_SIZE, NUM_IMAGES);

    dim3 threadsPerBlock_update_weights(UPDATE_WEIGHT_BLOCK_SIZE);
    dim3 blocksPerGrid_update_weights(IMG_SIZE, NUM_CLASSES);
    update_wieghts<<<blocksPerGrid_update_weights, threadsPerBlock_update_weights>>>(dt_images, dt_deltas, d_model.weights, learning_rate, NUM_IMAGES, NUM_CLASSES, IMG_SIZE);

    hipMemcpy(h_model.weights, d_model.weights, NUM_CLASSES * IMG_SIZE * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_model.biases, d_model.biases, NUM_CLASSES * sizeof(float), hipMemcpyDeviceToHost);

    print_model(h_model, input_size, NUM_CLASSES);
}